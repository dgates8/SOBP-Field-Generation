
#include <hip/hip_runtime.h>
#include <fstream>
#include <iterator>
#include <vector>
#include <iostream>
#include <cstdlib>
#include <string>
#include <sstream>
#include <iomanip>
#include <math.h>
#include <stdio.h>

void getSourceFile(std::vector<double>& eNomVec, std::vector<double>& rangeVec, 
			std::vector<double>& sigmaXVec,std::vector<double>& sigmaYVec, 
			std::vector<double>& eMeanVec, std::vector<double>& sigmaEVec, 
			std::vector<double>& xVec, std::vector<double>& yVec, 
			std::vector<double>& nxVec,std::vector<double>& nyVec,
			std::vector<double>& weightVec, int& numGroups) 
{
	int dateOfMeasurement;
	long int numberOfGroups;
	double  eNom, range, sigmaX, sigmaY, eMean, sigmaE, xcoord, ycoord, weight, nx, ny;
	
	std::string line;
	//declare and open file
	std::ifstream ifile("IMPT_source.dat", std::ios::in);
	if(!ifile){
		std::cout << "Error, IMPT_source not found" << std::endl;
	}else{
		//read in date of measurement
		ifile >> dateOfMeasurement;
	
		//read in number of groups
		ifile >> numberOfGroups;
		numGroups = numberOfGroups;
		
		//skip over header line
		std::string e, r, x, y, m, s, nx1, ny1, x1, y1, w;
		ifile >> e;
		ifile >> r;
		ifile >> x;
		ifile >> y;
		ifile >> m;
		ifile >> s;
		ifile >> x1;
		ifile >> y1;
		ifile >> nx1;
		ifile >> ny1;
		ifile >> w;

		eNomVec.reserve(numberOfGroups);
		
		//read in data to vectors
		for(int i = 0; i < numberOfGroups; i++){
			ifile >> eNom;
			ifile >> range;
			ifile >> sigmaX;
			ifile >> sigmaY;
			ifile >> eMean;
			ifile >> sigmaE;
			ifile >> xcoord;
			ifile >> ycoord;
			ifile >> nx;
			ifile >> ny;
			ifile >> weight;

			eNomVec.push_back(eNom);
//			rangeVec.push_back(range);
//			sigmaXVec.push_back(sigmaX);
//			sigmaYVec.push_back(sigmaY);
//			eMeanVec.push_back(eMean);
//			xVec.push_back(xcoord);
//			yVec.push_back(ycoord);
//			nxVec.push_back(nx);
//			nyVec.push_back(ny);
//			weightVec.push_back(weight);
		}
	}
}
int main(){
	int numberOfGroups;
	std::vector<double> eNom, range, sigmaX, sigmaY, eMean, sigmaE, xCoord, yCoord, nx, ny, weight;
	getSourceFile(eNom, range, sigmaX, sigmaY, eMean, sigmaE, xCoord, yCoord, nx, ny, weight, numberOfGroups);
	
	std::vector< std::vector<double> > zRange;
	
	for(int master = 0; master < 94; master++){
		
		std::vector<double> temp;

		//declare stream size variables and open file/check for errors
		std::streampos bufferSize;

		//create fileName to read in data
		std::ostringstream fName;
		if(master < 9){
			fName << std::fixed << "PercentEdep3D_field_0" << master+1 << "_" << std::setprecision(1) << eNom[master] << "MeV.bin";
		}else{
			fName << std::fixed << "PercentEdep3D_field_" << master+1 << "_"  << std::setprecision(1) << eNom[master] << "MeV.bin";

		}
		std::string fileName = fName.str();
		std::cout << fileName << std::endl;
		std::ifstream ifile(fileName.c_str(), std::ios::in | std::ios::binary);
		if(!ifile){
			std::cout << "Error, no file found" << std::endl;
			exit(1);
		}
		
		//get file size
		ifile.seekg(0, std::ios::end);
		bufferSize = ifile.tellg();
		ifile.seekg(0, std::ios::beg);

		//declare buffer
		std::vector<double> buffer(bufferSize/sizeof(double));
		
		//read in data
		ifile.read(reinterpret_cast<char*>(buffer.data()), bufferSize); 

		int size = bufferSize/(sizeof(double)*400);
		
		//copy memory from buffer to energy
		double *energy;
		energy = (double*)malloc(size*sizeof(double)*400);
		std::copy(buffer.begin(), buffer.end(), energy);

		for(int i = 20100; i < 16000000; i+= 40000){
			temp.push_back(energy[i]);
		}	 
		zRange.push_back(temp);		
	}
	
	std::cout << zRange.size() << std::endl;
	std::ofstream zfile("depthDose.txt", std::ios::out);
	zfile << "Depth Dose Curve" << std::endl;
	zfile << std::endl;
	zfile << "Z(cm)";
	for(int i = 0; i < 94; i++){
		zfile << std::fixed << std::setprecision(1) << std::setw(11) << eNom[i] << "(MeV)";
	}	
	zfile << std::endl;
	for(int i = 0; i < 400; i++){
		zfile << i << "        ";
		for(int j = 0; j < 94; j++){
			zfile << std::fixed << std::setprecision(3) << std::setw(8) << zRange[j][i] << "        ";
		}
		zfile << std::endl;
	}
	zfile << std::endl;
}

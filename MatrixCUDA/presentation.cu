#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <iostream> 
#include <iomanip>
#define speed 3.0e8
#define mass  0.511
#define hbar  1.68e-10
#define pi    3.1415
#define S     0.5 //Symmetry factor for two body event 
#define g     2.002319 //coupling constant for theory


/*************************************************************************
Handle-Error code for timing runs 
*************************************************************************/
static void HandleError(hipError_t err, const char *file, int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

/*************************************************************************
Device function to find energy using the standard formula 
*************************************************************************/
__device__ double Energy(int a){
	return sqrt(powf(a,2)*powf(speed,2) + powf(mass,2)*powf(speed,4));
}

/***************************************************************************
Scattering solves for the cross section of scattering for electron pair annhilation
e+ + e- -> y + y. No radial dependence so multiply by 4*pi. 

--Drake Gates example CUDA code
***************************************************************************/
__global__ void Scattering(int n, int *a, int *b, double *c){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n){
		double Etotal = 8*pi*(Energy(a[i])+Energy(b[i]));
		c[i] = powf((speed*hbar)/Etotal,2) * double(a[i])/double(b[i]) * 16*powf(g,4) * 4*pi;
	}
}	

int main(int argc, char* argv[]){
	int N = atoi(argv[1]);//Number of elements to be generated 
	int *v1, *v2, *d_v1, *d_v2;
	double *energy, *d_energy; //Declare vectors or matrices in row-max format

	//Allocate pointers of type int and length N 
	v1 = (int*)malloc(N*sizeof(int));
	v2 = (int*)malloc(N*sizeof(int));
	energy = (double*)malloc(N*sizeof(double));

	//random fill
	srand(time(NULL));
    for (int i = 0; i < N; i++){
    	v1[i] = mass*(rand() % 10) + 1;
    	v2[i] = mass*(rand() % 10) + 1;
    }

	//Allocate GPU pointers by reference and check errors 
	hipError_t err = hipMalloc(&d_v1, N*sizeof(int));
	printf("CUDA malloc v1: %s\n",hipGetErrorString(err));

	err = hipMalloc(&d_v2, N*sizeof(int));
	printf("CUDA malloc v2: %s\n",hipGetErrorString(err));

	err = hipMalloc(&d_energy, N*sizeof(double));
	printf("CUDA malloc energy: %s\n",hipGetErrorString(err));

	//copy vectors to GPU and check for errors
	err = hipMemcpy(d_v1, v1, N*sizeof(int), hipMemcpyHostToDevice);
	printf("Copy v1 to device: %s\n",hipGetErrorString(err));

	err= hipMemcpy(d_v2, v2, N*sizeof(int), hipMemcpyHostToDevice);
	printf("Copy v2 to device: %s\n",hipGetErrorString(err));

	//time the computation
	float time;
	hipEvent_t start, stop;

	HANDLE_ERROR( hipEventCreate(&start) );
	HANDLE_ERROR( hipEventCreate(&stop) );
	HANDLE_ERROR( hipEventRecord(start, 0) );

	//call kernel of 1 block with 10 threads  
	Scattering<<<(N+255)/256,256>>>(N, d_v1,d_v2,d_energy);
	//sync threads
	err = hipDeviceSynchronize();
	printf("Kernel Call: %s\n",hipGetErrorString(err));

	//get resultant energy off of device 
	err = hipMemcpy(energy, d_energy, N*sizeof(double), hipMemcpyDeviceToHost);
	printf("Copy Energy off device: %s\n",hipGetErrorString(err));

	HANDLE_ERROR( hipEventRecord(stop, 0) );
	HANDLE_ERROR( hipEventSynchronize(stop) );
	HANDLE_ERROR( hipEventElapsedTime(&time, start, stop) );
	printf("Time to generate:  %3.1f ms \n", time);

	//std::fixed << std::setprecision(6)
	/*
	for(int k = 0; k < N; k++){
		std::cout <<  energy[k] << "    " << v1[k] << "   " << v2[k] << std::endl;
	}
	*/
	//free GPU cache
	hipFree(d_v1);
	hipFree(d_v2);
	hipFree(d_energy);
	free(v1);
	free(v2);
	free(energy);

	return 0;
}
